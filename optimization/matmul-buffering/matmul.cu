#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

void naive_cpu_matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int k = 0; k < K; k++) {
      for (int j = 0; j < N; j++) {
        _C[i * N + j] += _A[i * K + k] * _B[k * N + j];
      }
    }
  }
}

#define BLOCKS 8
static int Mbegin[BLOCKS], Mend[BLOCKS];
static hipStream_t upload_stream, download_stream, calc_stream;
static hipEvent_t upload_events[BLOCKS], calc_events[BLOCKS];
static float *A_gpu, *B_gpu, *C_gpu;

__global__ void matmul_kernel(const float *A, const float *B, float *C, const int M, const int N, const int K) {
  // // if you change thx and thy then this will be faster
  const int tdix = blockIdx.x * blockDim.x + threadIdx.x;

  const int i = tdix / N;
  const int j = tdix % N;

  if (j >= N || i >= M) return;

  // init C
  C[i * N + j] = 0;

  float a0, a1, a2, a3, a4, a5, a6, a7;
  float b0, b1, b2, b3, b4, b5, b6, b7;
  int k;

  // loop unrolling
  for (k = 0; k + 7 < K; k += 8) {
    a0 = A[i * K + (k + 0)];
    a1 = A[i * K + (k + 1)];
    a2 = A[i * K + (k + 2)];
    a3 = A[i * K + (k + 3)];
    a4 = A[i * K + (k + 4)];
    a5 = A[i * K + (k + 5)];
    a6 = A[i * K + (k + 6)];
    a7 = A[i * K + (k + 7)];
    b0 = B[(k + 0) * N + j];
    b1 = B[(k + 1) * N + j];
    b2 = B[(k + 2) * N + j];
    b3 = B[(k + 3) * N + j];
    b4 = B[(k + 4) * N + j];
    b5 = B[(k + 5) * N + j];
    b6 = B[(k + 6) * N + j];
    b7 = B[(k + 7) * N + j];
    C[i * N + j] += a0 * b0 + a1 * b1 + a2 * b2 + a3 * b3 + a4 * b4 + a5 * b5 +
                    a6 * b6 + a7 * b7;
  }

  // Deal with trailing k
  for (; k < K; k++) { C[i * N + j] += A[i * K + k] * B[k * N + j]; }


}


void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  // Remove this line after you complete the matmul on GPU
    CHECK_CUDA(hipMemcpyAsync(B_gpu, _B, K * N * sizeof(float),
                             hipMemcpyHostToDevice, upload_stream));
  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipMemcpyAsync(&A_gpu[Mbegin[i] * K], &_A[Mbegin[i] * K],
                               (Mend[i] - Mbegin[i]) * K * sizeof(float),
                               hipMemcpyHostToDevice, upload_stream));
    CHECK_CUDA(hipEventRecord(upload_events[i], upload_stream));
  }

  for (int i = 0; i < BLOCKS; i++) {
    dim3 blockDim(1024);
    dim3 gridDim((N * (Mend[i] - Mbegin[i] ) + 1024 -1)/ 1024);

    CHECK_CUDA(hipStreamWaitEvent(calc_stream, upload_events[i]));
    matmul_kernel<<<gridDim, blockDim, 0, calc_stream>>>(
        &A_gpu[Mbegin[i] * K], B_gpu, &C_gpu[Mbegin[i] * N],
        (Mend[i] - Mbegin[i]), N, K);
    CHECK_CUDA(hipEventRecord(calc_events[i], calc_stream));
  }

  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipStreamWaitEvent(download_stream, calc_events[i]));
    CHECK_CUDA(hipMemcpyAsync(&_C[Mbegin[i] * N], &C_gpu[Mbegin[i] * N],
                               (Mend[i] - Mbegin[i]) * N * sizeof(float),
                               hipMemcpyDeviceToHost, download_stream));
  }
}

void matmul_init(int M, int N, int K) {
  for (int i = 0; i < BLOCKS; i++) {
    Mbegin[i] = M / BLOCKS * i;
    Mend[i] = M / BLOCKS * (i + 1);
    if (i == BLOCKS - 1) Mend[i] = M;
  }

  CHECK_CUDA(hipStreamCreate(&upload_stream));
  CHECK_CUDA(hipStreamCreate(&download_stream));
  CHECK_CUDA(hipStreamCreate(&calc_stream));
  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipEventCreate(&upload_events[i]));
    CHECK_CUDA(hipEventCreate(&calc_events[i]));
  }

  CHECK_CUDA(hipMalloc(&A_gpu, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_gpu, K * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_gpu, M * N * sizeof(float)));

}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));
  CHECK_CUDA(hipStreamDestroy(upload_stream));
  CHECK_CUDA(hipStreamDestroy(download_stream));
  CHECK_CUDA(hipStreamDestroy(calc_stream));
  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipEventDestroy(upload_events[i]));
    CHECK_CUDA(hipEventDestroy(calc_events[i]));
  }

}
