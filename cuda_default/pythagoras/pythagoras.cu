
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void pythagoras(int *pa, int *pb, int *pc, int *presult) {
  int a = *pa;
  int b = *pb;
  int c = *pc;

  if ((a * a + b * b) == c * c)
    *presult = 1;
  else
    *presult = 0;
}

int main(int argc, char *argv[]) {
  if (argc != 4) {
    printf("Usage: %s <num 1> <num 2> <num 3>\n", argv[0]);
    return 0;
  }

  int a = atoi(argv[1]);
  int b = atoi(argv[2]);
  int c = atoi(argv[3]);
  int result = 0;

  // TODO: 1. allocate device memory
  int *dev_a, *dev_b, *dev_c, *dev_result;
  CHECK_CUDA(hipMalloc(&dev_a, sizeof(int)));
  CHECK_CUDA(hipMalloc(&dev_b, sizeof(int)));
  CHECK_CUDA(hipMalloc(&dev_c, sizeof(int)));
  CHECK_CUDA(hipMalloc(&dev_result, sizeof(int)));


  // TODO: 2. copy data to device
  CHECK_CUDA(hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dev_c, &c, sizeof(int), hipMemcpyHostToDevice));

  // TODO: 3. launch kernel
  pythagoras<<<1,1>>>(dev_a, dev_b, dev_c, dev_result);


  // TODO: 4. copy result back to host
  CHECK_CUDA(hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(dev_a));
  CHECK_CUDA(hipFree(dev_b));
  CHECK_CUDA(hipFree(dev_c));
  CHECK_CUDA(hipFree(dev_result));

  if (result) printf("YES\n");
  else printf("NO\n");

  return 0;
}
