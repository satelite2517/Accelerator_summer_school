#include "hip/hip_runtime.h"
#include <cstdio>

#include "image_rotation.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *input_images_gpu, *output_images_gpu;

void rotate_image_naive(float *input_images, float *output_images, int W, int H,
                        float sin_theta, float cos_theta, int num_src_images) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  for (int i = 0; i < num_src_images; i++) {
    for (int dest_x = 0; dest_x < W; dest_x++) {
      for (int dest_y = 0; dest_y < H; dest_y++) {
        float xOff = dest_x - x0;
        float yOff = dest_y - y0;
        int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
        int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
        if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
          output_images[i * H * W + dest_y * W + dest_x] =
              input_images[i * H * W + src_y * W + src_x];
        } else {
          output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
        }
      }
    }
  }
}

__global__ void kernal_image_rotation(const float *input_image,float *output_image, const int W, const int H, const float sin_theta, const float cos_theta) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  int thx = blockIdx.x * blockDim.x + threadIdx.x;
  int thy = blockIdx.y * blockDim.y + threadIdx.y;

  float xOff = thx - x0;
  float yOff = thy - y0;

  int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
  int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);

  if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
    output_image[thy * W + thx] = input_image[src_y * W + src_x];
  } else {
    output_image[thy * W + thx] = 0.0f;
  }


}

void rotate_image(float *input_images, float *output_images, int W, int H,
                  float sin_theta, float cos_theta, int num_src_images) {
  // Remove this line after you complete the image rotation on GPU
  rotate_image_naive(input_images, output_images, W, H, sin_theta, cos_theta,
                     num_src_images);

  // (TODO) Upload input images to GPU
  for (int i = 0; i < num_src_images; i++) {
    CHECK_CUDA(hipMemcpy(input_images_gpu, input_images + i * W * H, sizeof(float) * W * H, hipMemcpyHostToDevice));
    dim3 block_dim(32, 32);
    dim3 grid_dim((W + block_dim.x - 1) / block_dim.x,
                  (H + block_dim.y - 1) / block_dim.y);
    kernal_image_rotation<<<grid_dim, block_dim>>>(input_images_gpu , output_images_gpu , W, H, sin_theta, cos_theta);
    CHECK_CUDA(hipMemcpy(output_images + i * W * H, output_images_gpu, sizeof(float) * W * H, hipMemcpyDeviceToHost));
  }
  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_init(int image_width, int image_height, int num_src_images) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&input_images_gpu, sizeof(float) * image_width * image_height));
  CHECK_CUDA(hipMalloc(&output_images_gpu, sizeof(float) * image_width * image_height ));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_cleanup() {
  // (TODO) Free device memory
  CHECK_CUDA(hipFree(input_images_gpu));
  CHECK_CUDA(hipFree(output_images_gpu));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
