#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void naive_cpu_matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int k = 0; k < K; k++) {
      for (int j = 0; j < N; j++) {
        _C[i * N + j] += _A[i * K + k] * _B[k * N + j];
      }
    }
  }
}

__global__ void matmal(const float *A, const float *B, float *C, const int M, const int N, const int K) {
  // if you change thx and thy then this will be faster
  int thx = blockIdx.x * blockDim.x + threadIdx.x;
  int thy = blockIdx.y * blockDim.y + threadIdx.y;
  if (thx>=M || thy>=N) return;

  C[thx * N + thy]= 0.0f;
  for (int i = 0; i <K; i++) {
    C[thx * N + thy] += A[thx * K + i] * B[i * N + thy];
  }

}

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  // Remove this line after you complete the matmul on GPU
  //naive_cpu_matmul(_A, _B, _C, M, N, K);

  // (TODO) Upload A and B matrix to GPU
  CHECK_CUDA(hipMemcpy(A_gpu, _A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(B_gpu, _B, K * N * sizeof(float), hipMemcpyHostToDevice));

  // (TODO) Launch kernel on a GPU
  dim3 block(32, 32);
  dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);
  matmal<<<grid, block>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  // (TODO) Download C matrix from GPU
  CHECK_CUDA(hipMemcpy(_C, C_gpu,M*N*sizeof(float), hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&A_gpu, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_gpu, K * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_gpu, M * N * sizeof(float)));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  free(_A)
  free(_B)
  free(_C)

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
