
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

int main() {
  int count;
  CHECK_CUDA(hipGetDeviceCount(&count));

  printf("Number of devices: %d\n", count);
  hipDeviceProp_t props[4];
  for (int i = 0; i < count; ++i) {
    printf("\tdevice %d:\n", i);

    // Fetch and print device properties
    CHECK_CUDA(hipGetDeviceProperties(&props[i], i));

    printf("  \tName: %s\n", props[i].name);
    printf("  \tSM count: %d\n", props[i].multiProcessorCount);
    printf("  \tMaxThreadsPerBlock: %d\n", props[i].maxThreadsPerBlock);
    printf("  \tTotal Global Memory: %lu bytes\n", props[i].totalGlobalMem);
    printf("  \tShared Memory Per Block: %lu bytes\n\n", props[i].sharedMemPerBlock);
  }

  return 0;
}
